/*Goal of Program
Create two arrays
-- I need an array that stays in the main as the header, h_
-- I need an array that goes into the thread/device, d_

Put the array into a thread
Call the Thread
The thread will only have one line of code, and that will be squaring itself
will it have to read itself? Will it matter? It should not matter
Have each piece of data in the thread cube itself
Free the memory
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
__global__
void cube(float *d_out, float *d_in) {
	int idx = threadIdx.x; 
	float f = d_in[idx]; 
	d_out[idx] = f * f * f; 
}
//Create the main file
int main() {
	//The size of the array
	const int arrSize = 50; 
	const int arrBytes = arrSize * sizeof(float);
	float h_in[arrSize], h_out[arrSize];
	float *d_in, *d_out; 
	//Allocate memory for d_in. Does not need & because d_in is already an address
	hipMalloc((void**) &d_in, arrBytes); 
	hipMalloc((void**) &d_out, arrBytes);
	//Initialize the h_in array, may remove later
	for (int i = 0; i < arrSize; i++) {
		h_in[i] = (float)i;
		
	}
	//Copy information from d_in to h_in
	hipMemcpy(d_in, h_in, arrBytes, hipMemcpyHostToDevice);
	cube <<<1, arrSize>>> (d_out, d_in);
	hipMemcpy(h_out, d_out, arrBytes, hipMemcpyDeviceToHost); 
	for (int i = 0; i < arrSize; i++) {
		printf("%f ", h_out[i]);
		printf((i % 4 == 3)?"\n":"\t"); 
	}
	hipFree(d_in);
	hipFree(d_out);

	
	return 0;

}